#include <stdio.h>

#include <warpKernels.cu>
#include <warpAlgorithms.hu>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void backwardWarp2D(const float* f,
                    const float* u,
                    const float* v,
                    float* fWarped,
                    int degree,
                    int width,
                    int height){

    size_t size = width*height * sizeof(float);

    // allocate vectors in device memory
    float *d_f, *d_u, *d_v, *d_fWarped;
    gpuErrchk(hipMalloc(&d_f, size));
    gpuErrchk(hipMalloc(&d_u, size));
    gpuErrchk(hipMalloc(&d_v, size));
    gpuErrchk(hipMalloc(&d_fWarped, size));

    // copy vectors from host memory to device memory
    gpuErrchk(hipMemcpy(d_f, f, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_u, u, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_v, v, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_fWarped, fWarped, size, hipMemcpyHostToDevice));

    // kernel invocation with 16*16 threads per block, and enough blocks
    // to cover the entire length of the vectors
    dim3 threadsPerBlock(16,16);
    dim3 numBlocks((width + 15)/16, (height + 15)/16);
    if(degree==1){
        linearBackwardWarp2DKernel<<<numBlocks, threadsPerBlock>>>(d_f,
                                                                   d_u,
                                                                   d_v,
                                                                   d_fWarped,
                                                                   width,
                                                                   height);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
    }else if(degree==3){
        float coeffs[] = {
            #include "cubic_2D_coefficients.inc"
        };
        float *d_coeffs;
        gpuErrchk(hipMalloc(&d_coeffs, 16*16*sizeof(float)));
        gpuErrchk(hipMemcpy(d_coeffs, coeffs, 16*16*sizeof(float), hipMemcpyHostToDevice));
        cubicBackwardWarp2DKernel<<<numBlocks, threadsPerBlock>>>(d_f,
                                                                  d_u,
                                                                  d_v,
                                                                  d_fWarped,
                                                                  width,
                                                                  height,
                                                                  d_coeffs);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
        hipFree(d_coeffs);
    }else{
        throw "Only degree 1 and 3 are implemented";
    }

    // copy the result back to the host
    gpuErrchk(hipMemcpy(fWarped, d_fWarped, size, hipMemcpyDeviceToHost));

    // release the device memory
    hipFree(d_f);
    hipFree(d_u);
    hipFree(d_v);
    hipFree(d_fWarped);
}


void backwardWarp3D(const float* f,
                    const float* u,
                    const float* v,
                    const float* w,
                    float* fWarped,
                    int degree,
                    int width,
                    int height,
                    int depth){

    size_t size = width*height*depth * sizeof(float);

    // allocate vectors in device memory
    float *d_f, *d_u, *d_v, *d_w, *d_fWarped;
    gpuErrchk(hipMalloc(&d_f, size));
    gpuErrchk(hipMalloc(&d_u, size));
    gpuErrchk(hipMalloc(&d_v, size));
    gpuErrchk(hipMalloc(&d_w, size));
    gpuErrchk(hipMalloc(&d_fWarped, size));

    // copy vectors from host memory to device memory
    gpuErrchk(hipMemcpy(d_f, f, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_u, u, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_v, v, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_w, w, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_fWarped, fWarped, size, hipMemcpyHostToDevice));

    // kernel invocation with 8*8*8 threads per block, and enough blocks
    // to cover the entire image
    dim3 threadsPerBlock(8,8,8);
    dim3 numBlocks((width + 7)/8, (height + 7)/8, (depth + 7)/8);
    if(degree==1){
        linearBackwardWarp3DKernel<<<numBlocks, threadsPerBlock>>>(d_f,
                                                                   d_u,
                                                                   d_v,
                                                                   d_w,
                                                                   d_fWarped,
                                                                   width,
                                                                   height,
                                                                   depth);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
    }else if(degree==3){
        float coeffs[] = {
            #include "cubic_3D_coefficients.inc"
        };
        float *d_coeffs;
        gpuErrchk(hipMalloc(&d_coeffs, 64*64*sizeof(float)));
        gpuErrchk(hipMemcpy(d_coeffs, coeffs, 64*64*sizeof(float), hipMemcpyHostToDevice));
        cubicBackwardWarp3DKernel<<<numBlocks, threadsPerBlock>>>(d_f,
                                                                  d_u,
                                                                  d_v,
                                                                  d_w,
                                                                  d_fWarped,
                                                                  width,
                                                                  height,
                                                                  depth,
                                                                  d_coeffs);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
        hipFree(d_coeffs);
    }else{
        throw "Only degree 1 and 3 are implemented";
    }

    // copy the result back to the host
    gpuErrchk(hipMemcpy(fWarped, d_fWarped, size, hipMemcpyDeviceToHost));

    // release the device memory
    hipFree(d_f);
    hipFree(d_u);
    hipFree(d_v);
    hipFree(d_w);
    hipFree(d_fWarped);
}


void adjointBackwardWarp2D(const float* fWarped,
                           const float* u,
                           const float* v,
                           float* f,
                           int degree,
                           int width,
                           int height){

    /*
    GPU implementation of 2D adjoint backward image warping along the DVF (u,v)
    with rectangular multivariate spline interpolation
    */


    size_t size = width*height * sizeof(float);

    // allocate vectors in device memory
    float *d_fWarped, *d_u, *d_v, *d_f;
    gpuErrchk(hipMalloc(&d_fWarped, size));
    gpuErrchk(hipMalloc(&d_u, size));
    gpuErrchk(hipMalloc(&d_v, size));
    gpuErrchk(hipMalloc(&d_f, size));

    // copy vectors from host memory to device memory
    gpuErrchk(hipMemcpy(d_fWarped, fWarped, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_u, u, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_v, v, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_f, f, size, hipMemcpyHostToDevice));

    // kernel invocation with 16*16 threads per block, and enough blocks
    // to cover the entire length of the vectors
    dim3 threadsPerBlock(16,16);
    dim3 numBlocks((width + 15)/16, (height + 15)/16);
    if(degree==1){
        adjointLinearBackwardWarp2DKernel<<<numBlocks, threadsPerBlock>>>(d_fWarped,
                                                                          d_u,
                                                                          d_v,
                                                                          d_f,
                                                                          width,
                                                                          height);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
    }else if(degree==3){
        adjointCubicBackwardWarp2DKernel<<<numBlocks, threadsPerBlock>>>(d_fWarped,
                                                                         d_u,
                                                                         d_v,
                                                                         d_f,
                                                                         width,
                                                                         height);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
    }else{
        throw "Only degree 1 and 3 are implemented";
    }

    // copy the result back to the host
    gpuErrchk(hipMemcpy(f, d_f, size, hipMemcpyDeviceToHost));

    // release the device memory
    hipFree(d_f);
    hipFree(d_u);
    hipFree(d_v);
    hipFree(d_fWarped);
}


void adjointBackwardWarp3D(const float* fWarped,
                           const float* u,
                           const float* v,
                           const float* w,
                           float* f,
                           int degree,
                           int width,
                           int height,
                           int depth){
    /*
    GPU implementation of 3D adjoint backward image warping along the DVF (u,v,w)
    with rectangular multivariate spline interpolation
    */

    size_t size = width*height*depth * sizeof(float);

    // allocate vectors in device memory
    float *d_fWarped, *d_u, *d_v, *d_w, *d_f;
    gpuErrchk(hipMalloc(&d_fWarped, size));
    gpuErrchk(hipMalloc(&d_u, size));
    gpuErrchk(hipMalloc(&d_v, size));
    gpuErrchk(hipMalloc(&d_w, size));
    gpuErrchk(hipMalloc(&d_f, size));

    // copy vectors from host memory to device memory
    gpuErrchk(hipMemcpy(d_fWarped, fWarped, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_u, u, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_v, v, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_w, w, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_f, f, size, hipMemcpyHostToDevice));

    // kernel invocation with 8*8*8 threads per block, and enough blocks
    // to cover the entire length of the vectors
    dim3 threadsPerBlock(8,8,8);
    dim3 numBlocks((width  + 8-1)/8,
                   (height + 8-1)/8,
                   (depth  + 8-1)/8);

    if(degree==1){
        adjointLinearBackwardWarp3DKernel<<<numBlocks, threadsPerBlock>>>(d_fWarped,
                                                                          d_u,
                                                                          d_v,
                                                                          d_w,
                                                                          d_f,
                                                                          width,
                                                                          height,
                                                                          depth);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
    }else if(degree==3){
        float coeffs[] = {
            #include "cubic_3D_coefficients.inc"
        };
        float *d_coeffs;
        gpuErrchk(hipMalloc(&d_coeffs, 64*64*sizeof(float)));
        gpuErrchk(hipMemcpy(d_coeffs, coeffs, 64*64*sizeof(float), hipMemcpyHostToDevice));
        adjointCubicBackwardWarp3DKernel<<<numBlocks, threadsPerBlock>>>(d_fWarped,
                                                                         d_u,
                                                                         d_v,
                                                                         d_w,
                                                                         d_f,
                                                                         width,
                                                                         height,
                                                                         depth,
                                                                         d_coeffs);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
        hipFree(d_coeffs);
    }else{
        throw "Only degree 1 and 3 are implemented";
    }
    // copy the result back to the host
    gpuErrchk(hipMemcpy(f, d_f, size, hipMemcpyDeviceToHost));

    // release the device memory
    hipFree(d_f);
    hipFree(d_u);
    hipFree(d_v);
    hipFree(d_w);
    hipFree(d_fWarped);
}