#include <stdio.h>

#include <warpKernelsAffine.cu>
#include <warpAlgorithmsAffine.hu>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void affineBackwardWarp2D(const float* f,
                          const float* A,
                          const float* b,
                          float* fWarped,
                          int degree,
                          int width,
                          int height){

    size_t size = width*height * sizeof(float);

    // allocate vectors in device memory
    float *d_f, *d_A, *d_b, *d_fWarped;
    gpuErrchk(hipMalloc(&d_f, size));
    gpuErrchk(hipMalloc(&d_A, 4 * sizeof(float)));
    gpuErrchk(hipMalloc(&d_b, 2 * sizeof(float)));
    gpuErrchk(hipMalloc(&d_fWarped, size));

    // copy vectors from host memory to device memory
    gpuErrchk(hipMemcpy(d_f, f, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_A, A, 4 * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_b, b, 2 * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_fWarped, fWarped, size, hipMemcpyHostToDevice));

    // kernel invocation with 16*16 threads per block, and enough blocks
    // to cover the entire length of the vectors
    dim3 threadsPerBlock(16,16);
    dim3 numBlocks((width + 15)/16, (height + 15)/16);
    if(degree==1){
        affineLinearBackwardWarp2DKernel<<<numBlocks, threadsPerBlock>>>(d_f,
                                                                         d_A,
                                                                         d_b,
                                                                         d_fWarped,
                                                                         width,
                                                                         height);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
    }else if(degree==3){
        // affineCubicBackwardWarp2DKernel<<<numBlocks, threadsPerBlock>>>(d_f,
        //                                                                 d_A,
        //                                                                 d_b,
        //                                                                 d_fWarped,
        //                                                                 width,
        //                                                                 height);
        // gpuErrchk(hipPeekAtLastError());
        // gpuErrchk(hipDeviceSynchronize());
        float coeffs[] = {
            #include "cubic_2D_coefficients.inc"
        };
        float *d_coeffs;
        gpuErrchk(hipMalloc(&d_coeffs, 16*16*sizeof(float)));
        gpuErrchk(hipMemcpy(d_coeffs, coeffs, 16*16*sizeof(float), hipMemcpyHostToDevice));
        affineCubicBackwardWarp2DKernel<<<numBlocks, threadsPerBlock>>>(d_f,
                                                                        d_A,
                                                                        d_b,
                                                                        d_fWarped,
                                                                        width,
                                                                        height,
                                                                        d_coeffs);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
        hipFree(d_coeffs);
    }else{
        throw "Only degree 1 and 3 are implemented";
    }

    // copy the result back to the host
    gpuErrchk(hipMemcpy(fWarped, d_fWarped, size, hipMemcpyDeviceToHost));

    // release the device memory
    hipFree(d_f);
    hipFree(d_A);
    hipFree(d_b);
    hipFree(d_fWarped);
}


void affineBackwardWarp3D(const float* f,
                          const float* A,
                          const float* b,
                          float* fWarped,
                          int degree,
                          int width,
                          int height,
                          int depth){

    size_t size = width*height*depth * sizeof(float);

    // allocate vectors in device memory
    float *d_f, *d_A, *d_b, *d_fWarped;
    gpuErrchk(hipMalloc(&d_f, size));
    gpuErrchk(hipMalloc(&d_A, 9 * sizeof(float)));
    gpuErrchk(hipMalloc(&d_b, 3 * sizeof(float)));
    gpuErrchk(hipMalloc(&d_fWarped, size));

    // copy vectors from host memory to device memory
    gpuErrchk(hipMemcpy(d_f, f, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_A, A, 9 * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_b, b, 3 * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_fWarped, fWarped, size, hipMemcpyHostToDevice));

    // kernel invocation with 8*8*8 threads per block, and enough blocks
    // to cover the entire image
    dim3 threadsPerBlock(8,8,8);
    dim3 numBlocks((width + 7)/8, (height + 7)/8, (depth + 7)/8);
    if(degree==1){
        affineLinearBackwardWarp3DKernel<<<numBlocks, threadsPerBlock>>>(d_f,
                                                                         d_A,
                                                                         d_b,
                                                                         d_fWarped,
                                                                         width,
                                                                         height,
                                                                         depth);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
    }else if(degree==3){
        float coeffs[] = {
            #include "cubic_3D_coefficients.inc"
        };
        float *d_coeffs;
        gpuErrchk(hipMalloc(&d_coeffs, 64*64*sizeof(float)));
        gpuErrchk(hipMemcpy(d_coeffs, coeffs, 64*64*sizeof(float), hipMemcpyHostToDevice));
        affineCubicBackwardWarp3DKernel<<<numBlocks, threadsPerBlock>>>(d_f,
                                                                        d_A,
                                                                        d_b,
                                                                        d_fWarped,
                                                                        width,
                                                                        height,
                                                                        depth,
                                                                        d_coeffs);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
        hipFree(d_coeffs);
    }else{
        throw "Only degree 1 and 3 are implemented";
    }

    // copy the result back to the host
    gpuErrchk(hipMemcpy(fWarped, d_fWarped, size, hipMemcpyDeviceToHost));

    // release the device memory
    hipFree(d_f);
    hipFree(d_A);
    hipFree(d_b);
    hipFree(d_fWarped);
}


void adjointAffineBackwardWarp2D(const float* fWarped,
                                 const float* A,
                                 const float* b,
                                 float* f,
                                 int degree,
                                 int width,
                                 int height){

    /*
    GPU implementation of 2D adjoint backward image warping along the DVF (u,v)
    with rectangular multivariate spline interpolation
    */


    size_t size = width*height * sizeof(float);

    // allocate vectors in device memory
    float *d_fWarped, *d_A, *d_b, *d_f;
    gpuErrchk(hipMalloc(&d_fWarped, size));
    gpuErrchk(hipMalloc(&d_A, 4 * sizeof(float)));
    gpuErrchk(hipMalloc(&d_b, 2 * sizeof(float)));
    gpuErrchk(hipMalloc(&d_f, size));

    // copy vectors from host memory to device memory
    gpuErrchk(hipMemcpy(d_fWarped, fWarped, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_A, A, 4 * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_b, b, 2 * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_f, f, size, hipMemcpyHostToDevice));

    // kernel invocation with 16*16 threads per block, and enough blocks
    // to cover the entire length of the vectors
    dim3 threadsPerBlock(16,16);
    dim3 numBlocks((width + 15)/16, (height + 15)/16);
    if(degree==1){
        adjointAffineLinearBackwardWarp2DKernel<<<numBlocks, threadsPerBlock>>>(d_fWarped,
                                                                                d_A,
                                                                                d_b,
                                                                                d_f,
                                                                                width,
                                                                                height);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
    }else if(degree==3){
        float coeffs[] = {
            #include "cubic_2D_coefficients.inc"
        };
        float *d_coeffs;
        gpuErrchk(hipMalloc(&d_coeffs, 16*16*sizeof(float)));
        gpuErrchk(hipMemcpy(d_coeffs, coeffs, 16*16*sizeof(float), hipMemcpyHostToDevice));
        adjointAffineCubicBackwardWarp2DKernel<<<numBlocks, threadsPerBlock>>>(d_fWarped,
                                                                               d_A,
                                                                               d_b,
                                                                               d_f,
                                                                               width,
                                                                               height,
                                                                               d_coeffs);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
        hipFree(d_coeffs);
    }else{
        throw "Only degree 1 and 3 are implemented";
    }

    // copy the result back to the host
    gpuErrchk(hipMemcpy(f, d_f, size, hipMemcpyDeviceToHost));

    // release the device memory
    hipFree(d_f);
    hipFree(d_A);
    hipFree(d_b);
    hipFree(d_fWarped);
}


void adjointAffineBackwardWarp3D(const float* fWarped,
                                 const float* A,
                                 const float* b,
                                 float* f,
                                 int degree,
                                 int width,
                                 int height,
                                 int depth){
    /*
    GPU implementation of 3D adjoint backward image warping along the DVF (u,v,w)
    with rectangular multivariate spline interpolation
    */

    size_t size = width*height*depth * sizeof(float);

    // allocate vectors in device memory
    float *d_fWarped, *d_A, *d_b, *d_f;
    gpuErrchk(hipMalloc(&d_fWarped, size));
    gpuErrchk(hipMalloc(&d_A, 9 * sizeof(float)));
    gpuErrchk(hipMalloc(&d_b, 3 * sizeof(float)));
    gpuErrchk(hipMalloc(&d_f, size));

    // copy vectors from host memory to device memory
    gpuErrchk(hipMemcpy(d_fWarped, fWarped, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_A, A, 9 * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_b, b, 3 * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_f, f, size, hipMemcpyHostToDevice));

    // kernel invocation with 8*8*8 threads per block, and enough blocks
    // to cover the entire length of the vectors
    dim3 threadsPerBlock(8,8,8);
    dim3 numBlocks((width  + 8-1)/8,
                   (height + 8-1)/8,
                   (depth  + 8-1)/8);

    if(degree==1){
        adjointAffineLinearBackwardWarp3DKernel<<<numBlocks, threadsPerBlock>>>(d_fWarped,
                                                                                d_A,
                                                                                d_b,
                                                                                d_f,
                                                                                width,
                                                                                height,
                                                                                depth);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
    }else if(degree==3){
        float coeffs[] = {
            #include "cubic_3D_coefficients.inc"
        };
        float *d_coeffs;
        gpuErrchk(hipMalloc(&d_coeffs, 64*64*sizeof(float)));
        gpuErrchk(hipMemcpy(d_coeffs, coeffs, 64*64*sizeof(float), hipMemcpyHostToDevice));
        adjointAffineCubicBackwardWarp3DKernel<<<numBlocks, threadsPerBlock>>>(d_fWarped,
                                                                               d_A,
                                                                               d_b,
                                                                               d_f,
                                                                               width,
                                                                               height,
                                                                               depth,
                                                                               d_coeffs);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
        hipFree(d_coeffs);
    }else{
        throw "Only degree 1 and 3 are implemented";
    }
    // copy the result back to the host
    gpuErrchk(hipMemcpy(f, d_f, size, hipMemcpyDeviceToHost));

    // release the device memory
    hipFree(d_f);
    hipFree(d_A);
    hipFree(d_b);
    hipFree(d_fWarped);
}